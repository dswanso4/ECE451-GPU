/*
Assignment: ECE 451 Programming Assignment 2
Code: GpuSumPrimes.cu
Group: David Swanson, Daniel Caballero, Michael Wilder

Description: This code adds all the prime numbers up to a certain number input by the user.
This code takes one parameter (SIZE) from the user and uses the CUDA library to run the 
calculations needed in parallel on 1024 threads.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 1024


/* SumPrimes the function on the device that calculates if a number is prime. It takes a 
pointer to the allocated array on the GPU and the size of the array. */

__global__ void SumPrimes (int *device_array, int SIZE) {
  
  // Index is calculated based on which block and thread is being worked.
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  int Prime = 1;

  // If the index is valid, then we need to check if it is prime.
  if (index < SIZE) {
    if ((index) == 0 || (index) == 1) {
      device_array[index] = 0;
	}

    // If the number is not prime, the value in the array is set to 0
	else {
      for (i=2; i*i <= index; i++) {
        if (index % i == 0) {
		  Prime = 0;
		  device_array[index] = 0;
		  break;
		}
	  }

      // if the number is prime, the value in the array is set to the number.
	  if (Prime)
		device_array[index] = index;
	}
  }
}

/* The main function of the code allocates memory on the host and device, transfers data
between the two, and calls the SumPrimes function. */

int main(int argc, char* argv []){
 
  int SIZE = atoi(argv[1]) + 1; 	
  int i;
  long int sum;
  int *host_array;
  int *device_array;
	
  sum = 0;
	
  // Allocate memory for host array and device array then copy host array to device array.
  host_array = (int *)malloc(SIZE*sizeof(int));
  hipMalloc(&device_array, SIZE*sizeof(int));
  hipMemcpy(device_array, host_array, SIZE*sizeof(int), hipMemcpyHostToDevice);

  // Define how many blocks and threads that need to be used when calling SumPrimes.
  // A 1D array is used. The size of blocksPerGrid is set in a way to prevent overflow.
  dim3 blocksPerGrid((SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE,1,1);
  dim3 threadsPerBlock(BLOCK_SIZE,1,1);
	
  SumPrimes <<<blocksPerGrid, threadsPerBlock>>>(device_array, SIZE);

  // Copy final array from device to host then clear memory in the device.
  hipMemcpy(host_array, device_array, SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipFree(device_array);

  // Testing print statement.
  printf("I am adding: ");
  
  // Add all the elements in the array. Only prime numbers will be non-zero.
  for (i = 0; i < SIZE; i++) {
    if (host_array[i] != 0)
	  printf("%d ", host_array[i]);
    sum += host_array[i];
  }

  printf("\nSum = %ld \n", sum);
	
  return 0;

}
